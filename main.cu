#include "hip/hip_runtime.h"
#include <iostream>
#include "inc/Timer.class.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "inc/Helper.class.h"
#include <thrust/transform_reduce.h>

using namespace std;
/**
 * Functor for applying to image vector
 */
struct copyelt2xp : public thrust::unary_function<int,int>
{
	int w, h, param;
    bool* m;
    bool* mxp;

    //==============================================================================
	//------------------------------------------------------------------------------
	// Author         Date        Action
	// S. Kondratov   05.02.2012  Implementation
	//==============================================================================
    copyelt2xp(bool* _m, bool* _mxp, int _w, int _h, int _param):
    m(_m), mxp(_mxp), w(_w), h(_h), param(_param)
	{
    }

    //==============================================================================
    // Algorithm implementation
    //------------------------------------------------------------------------------
    // Author         Date        Action
    // S. Kondratov   05.02.2012  Implementation
    //==============================================================================
    __device__
    int operator()(int i)
	{
        if (!m[i])
        {
	        int w_i = i % w;
	        int h_i = __float2int_rd(i / w);
	        int sum = 0;
	        
	        sum += chc(w_i - 1, w) && chc(h_i - 1, h) ? !m[i - 1 - w] : 1;
	        sum += chc(h_i - 1, h)                    ? !m[i - w]     : 1;
	        sum += chc(w_i + 1, w) && chc(h_i - 1, h) ? !m[i + 1 - w] : 1;
	        sum += chc(w_i - 1, w)                    ? !m[i - 1]     : 1;
	        sum += chc(w_i + 1, w) 		              ? !m[i + 1]     : 1;
	        sum += chc(w_i - 1, w) && chc(h_i + 1, h) ? !m[i - 1 + w] : 1;
	        sum += chc(h_i + 1, h)                    ? !m[i + w]     : 1;
	        sum += chc(w_i + 1, w) && chc(h_i + 1, h) ? !m[i + 1 + w] : 1;
	        
	        if (8 - sum > param)
	        {
	        	mxp[i] = true;
	    		return 1;
	        }
	    }
	    mxp[i] = m[i];
	    return 0;
    }
    //==============================================================================
	// Swap pointer to make new algorithm step
	//------------------------------------------------------------------------------
	// Author         Date        Action
	// S. Kondratov   05.02.2012  Implementation
	//==============================================================================
	__device__ __host__
    void Replace()
    {
    	bool* buff = m;
    	m = mxp;
    	mxp = buff;
    }

	//==============================================================================
	// Alias for logic functions
	//------------------------------------------------------------------------------
	// Author         Date        Action
	// S. Kondratov   05.02.2012  Implementation
	//==============================================================================
 	__device__
    inline bool chc(const int i, const int n)
    {
		return (n > i) && (i >= 0);
    }
};

//==============================================================================
// Function which implements main part of algorithm, which is ran on CPU
//------------------------------------------------------------------------------
// Author         Date        Action
// S. Kondratov   05.02.2012  Implementation
//==============================================================================
void StartAlgorithm(thrust::host_vector<bool>& m, const int width, const int height, const int param)
{
	// Allocate memory to use pointer arithmetic
	bool* mat = (bool*)malloc(width * height * sizeof(bool));
	bool* matxp = (bool*)malloc(width * height * sizeof(bool));
	
	// Create vectors on allocated memory
	thrust::device_vector<bool> devVec(mat, mat + width * height);
	thrust::device_vector<bool> devVecResult(matxp, matxp + width * height);

	// Fill vector
	devVec.assign(m.begin(), m.end());
	
	thrust::counting_iterator<int>seqb(0);
	thrust::counting_iterator<int>seqe = seqb + width * height;
	
	// Is that save iterate through raw pointer,
	// which allocated using advanced allocator..?
	thrust::device_vector<bool>::iterator m_input = devVec.begin();
	thrust::device_vector<bool>::iterator m_output = devVecResult.begin();

	copyelt2xp functor(thrust::raw_pointer_cast(&m_input[0]),
					   thrust::raw_pointer_cast(&m_output[0]),
					   width,
					   height,
					   param);
	int repeats = 0;
	int res = 1;
	while (res)
	{
		res = thrust::transform_reduce(seqb, seqe, functor, (int)0, thrust::plus<int>());
		functor.Replace();
		if( ++repeats == 150) break;
	}

	if (repeats % 2)
	{
		m.assign(devVec.begin(), devVec.end());
	}
	else
	{
		m.assign(devVecResult.begin(), devVecResult.end());
	}
}

int main( int argc, char* argv[] )
{
	CalcTimer timer1;

	if( argc != 4 )
	{
	  cout << "Usage: imgsmth <input_filename> <output_filename> <param_value>"
	  << endl << endl;
	  return 1;
	}

	// declare and read the bitmap
	BMP Input;
	Input.ReadFromFile(argv[1]);
	thrust::host_vector<bool> result(0);
	Helper::MkeMonochrome(Input);
	Helper::CreateHostVectorFromImage(result, Input);

	StartAlgorithm(result, Input.TellWidth(), Input.TellHeight(), atoi(argv[3]));

	Helper::CreateImageFromHostVector(result, Input);
	Input.WriteToFile(argv[2]);
	return 0;
}




